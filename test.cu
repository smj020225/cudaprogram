
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
	printf("hello GPU\n");
}

int main(void){
	hello<<<4,4>>>();
	hipDeviceSynchronize();

	return 0;
}
