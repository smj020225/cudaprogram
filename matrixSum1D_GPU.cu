#include "hip/hip_runtime.h"
#include <stdio.h>

#include "tool/common.cuh"

__global__ void addFromGPU(float *A, float *B, float *C, const int n){
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id = tid + bid * blockDim.x;

    C[id] = A[id] + B[id];
}

void initialData(float *addr, int elemCount){
    for (int i = 0; i < elemCount; i++)
        addr[i] = (float)(rand() & 0xFF) / 10.f;
    return;
}

int main(void){
    setGpu();
    
    // 分配主机内存与设备内存
    int iElemCount = 512;
    size_t stBytesCount = iElemCount * sizeof(float);

    // 分配主机内存并初始化
    float *fpHost_A, *fpHost_B, *fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if(fpHost_A!=NULL && fpHost_B!=NULL && fpHost_C!=NULL){
        memset(fpHost_A, 0, stBytesCount);
        memset(fpHost_B, 0, stBytesCount);
        memset(fpHost_C, 0, stBytesCount);
    } else {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }

    // 分配设备内存，并初始化
    float *fpDevice_A, *fpDevice_B, *fpDevice_C;
    hipMalloc((float **)&fpDevice_A, stBytesCount);
    hipMalloc((float **)&fpDevice_B, stBytesCount);
    hipMalloc((float **)&fpDevice_C, stBytesCount);
    if(fpDevice_A!=NULL && fpDevice_B!=NULL && fpDevice_C!=NULL){
        hipMemset(fpDevice_A, 0, stBytesCount);
        hipMemset(fpDevice_B, 0, stBytesCount);
        hipMemset(fpDevice_C, 0, stBytesCount);
    } else {
        printf("Fail to allocate memory!\n");
        free(fpHost_A);
        free(fpHost_B);
        free(fpHost_C);
        exit(-1);
    }
    

    // 初始化主机中的数据
    srand(666);
    initialData(fpHost_A, iElemCount);
    initialData(fpHost_B, iElemCount);

    // 数据从主机复制到设备
    hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice);
    hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice);
    hipMemcpy(fpDevice_C, fpHost_C, stBytesCount, hipMemcpyHostToDevice);

    // 调用和函数在设备中进行计算
    dim3 block(32);
    dim3 grid(iElemCount / 32);

    addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C, iElemCount);
    hipDeviceSynchronize();

    // 将计算得到的数据从设备传给主机
    hipMemcpy(fpHost_C, fpDevice_C, stBytesCount, hipMemcpyDeviceToHost);

    for (int i=0; i<10; i++){
        printf("idx%2d\tmatrix_a:%.2f\tmatrix_B:%.2f\tresult=%.2f\n", i+1, fpHost_A[i], fpHost_B[i], fpHost_C[i]);
    }

    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    hipFree(fpDevice_A);
    hipFree(fpDevice_B);
    hipFree(fpDevice_C);

    hipDeviceReset();
    return 0;
}